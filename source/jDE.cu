#include "hip/hip_runtime.h"
#include "jDE.cuh"

jDE::jDE( uint _s, uint _ndim, float _x_min, float _x_max ):
  NP(_s),
  n_dim(_ndim),
  x_min(_x_min),
  x_max(_x_max)
{
  checkCudaErrors(hipMalloc((void **)&F,  NP * sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&CR, NP * sizeof(float)));
  thrust::fill(thrust::device, F , F  + NP, 0.50);
  thrust::fill(thrust::device, CR, CR + NP, 0.90);

  checkCudaErrors(hipMalloc((void **)&T_F,  NP * sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&T_CR, NP * sizeof(float)));
  thrust::fill(thrust::device, T_F , T_F  + NP, 0.50);
  thrust::fill(thrust::device, T_CR, T_CR + NP, 0.90);

  Configuration conf;
  conf.x_min = x_min;
  conf.x_max = x_max;
  conf.ps = NP;
  conf.n_dim = n_dim;

  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(params), &conf, sizeof(Configuration)));
  checkCudaErrors(hipMalloc((void **)&rseq, NP * sizeof(uint)));
  checkCudaErrors(hipMalloc((void **)&fseq, 3 * NP * sizeof(uint)));
  checkCudaErrors(hipMalloc((void **)&d_states, NP * sizeof(hiprandStateXORWOW_t)));
  thrust::sequence(thrust::device, rseq, rseq + NP);

  NT_A.x = 32;
  NB_A.x = (NP%32)? (NP/32)+1 : NP/32;

  NT_B.x = 32 * ceil((double) n_dim / 32.0);
  NB_B.x = NP;

  std::random_device rd;
  unsigned int seed = rd();
  setup_kernel<<<NT_A, NB_A>>>(d_states, seed);
  checkCudaErrors(hipGetLastError());

  checkCudaErrors(hipMalloc((void **)&d_states2, NP * n_dim * sizeof(hiprandStateXORWOW_t)));
  sk2<<<NB_B, NT_B>>>(d_states2, seed);
  checkCudaErrors(hipGetLastError());
}

jDE::~jDE()
{
  checkCudaErrors(hipFree(F));
  checkCudaErrors(hipFree(CR));
  checkCudaErrors(hipFree(T_F));
  checkCudaErrors(hipFree(T_CR));
  checkCudaErrors(hipFree(rseq));
  checkCudaErrors(hipFree(fseq));
  checkCudaErrors(hipFree(d_states));
  checkCudaErrors(hipFree(d_states2));
}

void jDE::reset(){
  thrust::fill(thrust::device, F , F  + NP, 0.50);
  thrust::fill(thrust::device, CR, CR + NP, 0.90);

  thrust::fill(thrust::device, T_F , T_F  + NP, 0.50);
  thrust::fill(thrust::device, T_CR, T_CR + NP, 0.90);
}

void jDE::update(){
  updateK<<<NB_A, NT_A>>>(d_states, F, CR, T_F, T_CR);
  checkCudaErrors(hipGetLastError());
}


/*
 * fog == fitness of the old offspring
 * fng == fitness of the new offspring
 */
void jDE::run(float * og, float * ng){
  rand_DE<<<NB_B, NT_B>>>(d_states2, og, ng, T_F, T_CR, fseq);
  checkCudaErrors(hipGetLastError());
}

void jDE::run_b(float * og, float * ng, float * bg, float * fog, float * fng, uint b_id){
  best_DE<<<NB_B, NT_B>>>(og, ng, bg, fog, fng, b_id);
  checkCudaErrors(hipGetLastError());
}

void jDE::index_gen(){
  iGen<<<NB_A, NT_A>>>(d_states, rseq, fseq);
  checkCudaErrors(hipGetLastError());
}

void jDE::selection(float * og, float * ng, float * fog, float * fng){
  selectionK<<<NB_A, NT_A>>>(og, ng, fog, fng);
  checkCudaErrors(hipGetLastError());
}

void jDE::crowding_selection(
  float * og, float * ng,
  float * fog, float * fng,
  float * res
){
  float * iter;
  int position;
  thrust::device_ptr<float> d_fog = thrust::device_pointer_cast(fog);
  thrust::device_ptr<float> d_fng = thrust::device_pointer_cast(fng);

  thrust::device_ptr<float> d_f = thrust::device_pointer_cast(F);
  thrust::device_ptr<float> d_cr = thrust::device_pointer_cast(CR);

  thrust::device_ptr<float> d_tf = thrust::device_pointer_cast(T_F);
  thrust::device_ptr<float> d_tcr = thrust::device_pointer_cast(T_CR);
  // thrust::device_ptr<float> d_og = thrust::device_pointer_cast(og);
  // thrust::device_ptr<float> d_ng = thrust::device_pointer_cast(ng);

  // printf("NP: %d\n", NP);
  for( uint p = 0; p < NP; p++ ){
    crowding<<<NB_A, NT_A>>>(ng, og, p, res);
    checkCudaErrors(hipGetLastError());
    // thrust::device_vector<float> d_res( res, res+NP);
    // thrust::host_vector<float> h_res = d_res;
    // for( int i = 0 ; i < NP; i++ ){
    //   printf("%.1f ", h_res[i]);
    // }
    // printf("\n");
    iter = thrust::min_element(thrust::device, res, res + NP);
    position = iter - res;
    //
    // printf("[%d] The minimum distance element is: %i %.3f\n", p+1, position, h_res[position]);
    // printf("d_fng[%d] %.4f <= d_fog[%d] %.4f\n", p, (float)d_fng[p], position, (float)d_fog[position]);
    if( (float)d_fng[p] <= (float)d_fog[position] ){
      // printf("Antes: %.3f e \n", (float)d_fog[position]);
      // for( int i = 0; i < n_dim; i++ ){
      //   printf("%.1f ", (float)d_og[position * n_dim + i]);
      // }
      // printf("\n");

      thrust::copy_n(thrust::device,
        ng + (p * n_dim),       //source
        n_dim,                  //num elements to copy
        og + (position * n_dim) //destination
      );

      thrust::copy_n(thrust::device,
        fng + p,       //source fitness
        1,             //copy just one value
        fog + position //destination fitness update
      );

      // update F and CR;
      d_f[position]  = d_tf[p];
      d_cr[position] = d_tcr[p];

      // printf("Agora: %.3f e \n", (float)d_fog[position]);
      // for( int i = 0; i < n_dim; i++ ){
      //   printf("%.1f ", (float)d_og[position * n_dim + i]);
      // }
      // printf("\n");
      //
      // printf("E deve ser: \n");
      // for( int i = 0; i < n_dim; i++ ){
      //   printf("%.1f ", (float)d_ng[p * n_dim + i]);
      // }
      // printf("\n");

    }
  }
  // scanf("%d", &position);
}

/*
 * Update F and CR values accordly with jDE algorithm.
 *
 * F_Lower, F_Upper and T are constant variables declared
 * on constants header
 */
__global__ void updateK(hiprandState * g_state, float * d_F, float * d_CR, float * d_TF, float * d_TCR) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;

  uint ps = params.ps;

  if( index < ps ){
    hiprandState localState;
    localState = g_state[index];

    //(0, 1]
    float r1, r2, r3, r4;
    r1 = hiprand_uniform(&localState);
    r2 = hiprand_uniform(&localState);
    r3 = hiprand_uniform(&localState);
    r4 = hiprand_uniform(&localState);

    if (r2 < T){
      d_TF[index] = F_Lower + (r1 * F_Upper);
    } else {
      d_TF[index] = d_F[index];
    }

    if (r4 < T){
      d_TCR[index] = r3;
    } else {
      d_TCR[index] = d_CR[index];
    }

    g_state[index] = localState;
  }
}

/*
 * Performs the selection step
 * In this case, each thread is a individual
 * og -> Old genes, the previous generation offspring
 * ng -> New genes, the new generation offsprings
 * fog -> fitness of the old offspring
 * fng -> fitness of the new offspring
 */
__global__ void selectionK(float * og, float * ng, float * fog, float * fng){
  uint index = threadIdx.x + blockDim.x * blockIdx.x;
  uint ps = params.ps;

  if( index < ps ){
    uint ndim = params.n_dim;
    if( fng[index] <= fog[index] ){
      memcpy(og + (ndim * index), ng + (ndim * index), ndim * sizeof(float));
      fog[index]  = fng[index];
   }
  }
}

/*
 * Performs the DE/rand/1/bin operation
 * 1 thread == 1 individual
 * rng == global random state
 * fog == fitness of the old offspring
 * fng == fitness of the new offspring
 * F == mutation factor vector
 * CR == crossover probability vector
 */
__global__ void DE(hiprandState * rng, float * og, float * ng, float * F, float * CR, uint * fseq){
  uint i, index, ps, n_dim;
  index = threadIdx.x + blockDim.x * blockIdx.x;
  ps = params.ps;

  if(index < ps){
    uint n1, n2, n3, p1, p2, p3, p4;
    n_dim = params.n_dim;

    float mF  = F[index];
    float mCR = CR[index];

    hiprandState random = rng[index];

    n1 = fseq[index];
    n2 = fseq[index + ps];
    n3 = fseq[index + ps + ps];

    //do n1 = hiprand(&random)%ps; while (n1 == index);
    //do n2 = hiprand(&random)%ps; while (n2 == index || n2 == n1 );
    //do n3 = hiprand(&random)%ps; while (n3 == index || n3 == n1 || n3 == n2);

    p1 = index * n_dim;
    p2 = n3 * n_dim;
    p3 = n2 * n_dim;
    p4 = n1 * n_dim;
    //printf("[%u] %u %u %u => %u %u %u %u\n", index, n1, n2, n3, p4, p3, p2, p1);
    for( i = 0; i < n_dim; i++ ){
      if( hiprand_uniform(&random) <= mCR || (i == n_dim - 1) ){
        /* Get three mutually different indexs */
        ng[p1 + i] = og[p2 + i] + mF * (og[p3 + i] - og[p4 + i]);

        /* Check bounds */
        ng[p1 + i] = max(params.x_min, ng[p1 + i]);
        ng[p1 + i] = min(params.x_max, ng[p1 + i]);
      } else {
        ng[p1 + i] = og[p1 + i];
      }
    }
    rng[index] = random;
  }
}

__global__ void rand_DE(hiprandState *rng, float * og, float * ng, float * F, float * CR, uint * fseq){
  uint id_d, id_p, ps, n_dim;

  //id_g = threadIdx.x + blockDim.x * blockIdx.x;

  id_d = blockIdx.x;
	id_p = threadIdx.x;

  n_dim = params.n_dim;
  ps = params.ps;

  __syncthreads();

  if( id_p < n_dim ){
    hiprandState random = rng[ id_d * id_p ];

    __shared__ uint n1, n2, n3, p1, p2, p3, p4, rnbr;
    __shared__ float mF, mCR;

    if( id_p == 0 ){
      n1 = fseq[id_d];
      n2 = fseq[id_d + ps];
      n3 = fseq[id_d + ps + ps];

      mF  = F[id_d];
      mCR = CR[id_d];

      p1 = id_d * n_dim;
      p2 = n3 * n_dim;
      p3 = n2 * n_dim;
      p4 = n1 * n_dim;

      rnbr = hiprand(&random) % n_dim;
    }

    __syncthreads();

    if( hiprand_uniform(&random) <= mCR || (id_p == rnbr) ){
      ng[p1 + id_p] = og[p2 + id_p] + mF * (og[p3 + id_p] - og[p4 + id_p]);

      // ng[p1 + id_p] = max(params.x_min, ng[p1 + id_p]);
      // ng[p1 + id_p] = min(params.x_max, ng[p1 + id_p]);
      if( ng[p1 + id_p] <= params.x_min ){
        ng[p1 + id_p] += 2.0 * params.x_max;
      } else if( ng[p1 + id_p] > params.x_max ){
        ng[p1 + id_p] += 2.0 * params.x_min;
      }
    } else {
      ng[p1 + id_p] = og[p1 + id_p];
    }

    rng[id_d * id_p ] = random;
  }
}

__global__ void best_DE(float * og, float * ng, float * bnew, float * fog, float * fng, uint pbest){
  uint id_d, id_p, n_dim;

  //id_g = threadIdx.x + blockDim.x * blockIdx.x;

  id_d  = blockIdx.x;
	id_p  = threadIdx.x;

  n_dim = params.n_dim;

  if( id_p < n_dim ){
    __shared__ uint p1;
    __shared__ uint pb;

    __shared__ float _FA;
    __shared__ float _FB;

    __syncthreads();

    if( id_p == 0 ){
      p1 = id_d * n_dim;
      pb = pbest * n_dim;

      _FA = fog[id_d];
      _FB = fng[id_d];
    }

    __syncthreads();

    // if(id_p == 0 && id_d == 0){
    //   for(int i = 0; i < n_dim; i++){
    //     printf("teste[%d] = %.3f;\n", i, og[pb + i]);
    //   }
    // }

    if( _FB <= _FA ){
      bnew[p1 + id_p] = og[pb + id_p] + 0.5 * (ng[p1 + id_p] - og[p1 + id_p]);

      //check bounds
      if( bnew[p1 + id_p] <= params.x_min ){
        bnew[p1 + id_p] += 2.0 * params.x_max;
      } else if( bnew[p1 + id_p] > params.x_max ){
        bnew[p1 + id_p] += 2.0 * params.x_min;
      }
    } else {
      bnew[p1 + id_p] = ng[p1 + id_p];
    }
  }
}

/*
 * Performs the crowding operation
 * One thread per individual
 *
 * The kernel calculate the projection of a vector (A)
 * in a set of vectors (B)
 *
 * @params:
 * float * A: fixed vector to compare
 * float * B: array to compare
 * uint pid: the fixed index to compare
 * float * res: stores the distance;
 */
__global__ void crowding(float * A, float * B, uint p_id, float * res){
  uint t_id = threadIdx.x + blockDim.x * blockIdx.x;

  uint N  = params.n_dim;
  uint PS = params.ps;

  if( t_id < PS ){
    float S = 0.0; //stores the sum
    float D = 0.0; //stores the distance

    for( uint i = 0; i < N; i++ ){
      D = A[p_id * N + i] - B[t_id * N + i];
      S += D * D;
    }

    res[t_id] = S;
  }

}

/*
 * Generate 3 different indexs to DE/rand/1/bin.
 * @TODO:
 *  + rseq on constant memory;
 */
__global__ void iGen(hiprandState * g_state, uint * rseq, uint * fseq){
  uint index = threadIdx.x + blockDim.x * blockIdx.x;

  uint ps = params.ps;
  if( index < ps ){
    hiprandState s = g_state[index];

    uint n1, n2, n3;

    n1 = hiprand(&s) % ps;
    if( rseq[n1] == index )
      n1 = (n1 + 1) % ps;

    n2 = ( hiprand(&s) % ((int)ps/3) ) + 1;
    if( rseq[(n1 + n2) % ps] == index )
      n2 = (n2 + 1) % ps;

    n3 = ( hiprand(&s) % ((int)ps/3) ) + 1;
    if( rseq[(n1 + n2 + n3) % ps] == index )
      n3 = (n3 + 1 ) % ps;

    fseq[index] = rseq[n1];
    fseq[index+ps] = rseq[(n1+n2)%ps];
    fseq[index+ps+ps] = rseq[(n1+n2+n3)%ps];

    g_state[index] = s;
    //printf("[%-3d] %-3d | %-3d | %-3d\n", index, rseq[n1], rseq[(n1+n2)%ps], rseq[(n1+n2+n3)%ps]);
  }
}

/* Each thread gets same seed, a different sequence number, no offset */
__global__ void setup_kernel(hiprandState * random, uint seed){
  uint index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index < params.ps)
    hiprand_init(seed, index, 0, &random[index]);
}

/*
 *
 * Setup kernel version 2
 *
 */
__global__ void sk2(hiprandState * random, uint seed){
  uint index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index < params.ps * params.n_dim)
    hiprand_init(seed, index, 0, &random[index]);
}
